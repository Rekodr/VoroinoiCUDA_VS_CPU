#include <vector>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>




#define BLOCKSIZE_x 32
#define BLOCKSIZE_y 32

using namespace std;


__device__ double eucludianDist(int Ax, int Ay, int Bx, int By) {
	double d = sqrt(pow((Ax - Bx), 2) +
									pow((Ay - By), 2)
							);
	return d;
}

__global__ void voronoiKernel(int* vec, int cols, int rows, int pitch, int* Px, int* Py, int numSeeds) {
	  int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;

		int* row_a = (int*)((char*)vec + y * pitch) + x;

		__syncthreads();
    if((x < cols) && (y < rows)){
      double d = eucludianDist(x, y, Px[0], Py[0]);
			__syncthreads();
			//atomicExch(row_a, 0);
			*row_a = 0;
      for (int i = 0; i < numSeeds ; i++) {
				double temp = eucludianDist(x, y, Px[i], Py[i]);
				__syncthreads();
        if (temp < d) {
          d = temp;
					*row_a = i;
        }
      }
    }
}

void voronoi_cuda(int* h_vec, std::vector<std::pair<unsigned int, unsigned int>> &P, int cols = 0, int rows = 0) {

  hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	int* vec_dev;
	int* Px_dev;
	int* Py_dev;
	size_t pitch;
	int nSeeds = P.size();
  int ln = nSeeds * sizeof(int) ;

	int* Px = (int*) malloc(ln);
	int* Py = (int*) malloc(ln);

	int i = 0;
	// could not figure an easy way to copy vector of pair to the gpu
	for(auto& p : P){
		Px[i] = p.first;
		Py[i] = p.second;
		i++;
	}

 std::cout << "computing veronoi->Parallel..." << std::endl;
	hipMallocPitch((void**)&vec_dev, &pitch, sizeof(int)*cols, rows);

	hipEventRecord(start); //start counting event on GPU

	hipMalloc((void**)&Px_dev, ln);
	hipMalloc((void**)&Py_dev, ln);
	hipMemcpy(Px_dev, Px, ln, hipMemcpyHostToDevice);
	hipMemcpy(Py_dev, Py, ln, hipMemcpyHostToDevice);

	dim3 blockDim(BLOCKSIZE_y, BLOCKSIZE_x);
	dim3 gridDim(ceil((float)cols /blockDim.x) , ceil((float)rows /blockDim.y));


	voronoiKernel<<<gridDim, blockDim>>>(vec_dev, cols, rows, pitch, Px_dev, Py_dev, nSeeds);

	hipMemcpy2D(h_vec, sizeof(int)*cols, vec_dev, pitch, sizeof(int)*cols, rows, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	hipEventRecord(end);
	hipEventSynchronize(end);
	std::cout << "done..." << std::endl;
  float ms{0};
	hipEventElapsedTime(&ms, start, end);
	std::cout << "computation time(ms): " << ms << std::endl;
	hipFree(vec_dev);
	hipFree(Px_dev);
	hipFree(Py_dev);
	free(Px);
	free(Py);

}
